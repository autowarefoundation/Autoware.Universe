#include "hip/hip_runtime.h"
// Copyright 2022 Tier IV, Inc.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <nms_kernel.hpp>
#include <postprocess_kernel.hpp>

#include <thrust/count.h>
#include <thrust/sort.h>

namespace centerpoint
{
struct is_not_suppressed
{
  __device__ bool operator()(const Box3D & b) { return !b.is_suppressed; }
};

struct score_greater
{
  __device__ bool operator()(const Box3D & lb, const Box3D & rb) { return lb.score > rb.score; }
};

struct is_kept
{
  __device__ bool operator()(const bool keep) { return keep; }
};

__device__ float sigmoid(float x) { return 1.0f / expf(-x); }

__global__ void generateBoxes3D_kernel(
  const float * out_heatmap, const float * out_offset, const float * out_z, const float * out_dim,
  const float * out_rot, const float * out_vel, const float voxel_size_x, const float voxel_size_y,
  const float range_min_x, const float range_min_y, const int down_grid_size_x,
  const int down_grid_size_y, const int downsample_factor, const int num_class,
  const float score_threshold, Box3D * det_boxes3d)
{
  // generate boxes3d from the outputs of the network.
  // shape of out_*: (N, DOWN_GRID_SIZE_Y, DOWN_GRID_SIZE_X)
  // heatmap: N = num_class, offset: N = 2, z: N = 1, dim: N = 3, rot: N = 2, vel: N = 2
  int yi = blockIdx.x;
  int xi = threadIdx.x;
  int idx = down_grid_size_x * yi + xi;
  int down_grid_size = down_grid_size_y * down_grid_size_x;

  int label = -1;
  float max_score = -1;
  for (int ci = 0; ci < num_class; ci++) {
    float score = sigmoid(out_heatmap[down_grid_size * ci + idx]);
    if (score > max_score) {
      label = ci;
      max_score = score;
    }
  }

  float offset_x = out_offset[down_grid_size * 0 + idx];
  float offset_y = out_offset[down_grid_size * 1 + idx];
  float x = voxel_size_x * downsample_factor * (xi + offset_x) + range_min_x;
  float y = voxel_size_y * downsample_factor * (yi + offset_y) + range_min_y;
  float z = out_z[idx];
  float w = out_dim[down_grid_size * 0 + idx];
  float l = out_dim[down_grid_size * 1 + idx];
  float h = out_dim[down_grid_size * 2 + idx];
  float rot_sin = out_rot[down_grid_size * 0 + idx];
  float rot_cos = out_rot[down_grid_size * 1 + idx];
  float theta = atan2f(rot_sin, rot_cos);
  float vel_x = out_vel[down_grid_size * 0 + idx];
  float vel_y = out_vel[down_grid_size * 1 + idx];

  det_boxes3d[idx].label = label;
  det_boxes3d[idx].score = max_score;
  det_boxes3d[idx].x = x;
  det_boxes3d[idx].y = y;
  det_boxes3d[idx].z = z;
  det_boxes3d[idx].length = expf(l);
  det_boxes3d[idx].width = expf(w);
  det_boxes3d[idx].height = expf(h);
  det_boxes3d[idx].yaw = atan2f(rot_sin, rot_cos);
  det_boxes3d[idx].vel_x = vel_x;
  det_boxes3d[idx].vel_y = vel_y;
  det_boxes3d[idx].is_suppressed = max_score < score_threshold;
}

PostProcessCUDA::PostProcessCUDA(const int num_class) : num_class_(num_class)
{
  const int num_raw_boxes3d = Config::down_grid_size_y * Config::down_grid_size_x;
  boxes3d_d_ = thrust::device_vector<Box3D>(num_raw_boxes3d);
}

hipError_t PostProcessCUDA::generateDetectedBoxes3D_launch(
  const float * out_heatmap, const float * out_offset, const float * out_z, const float * out_dim,
  const float * out_rot, const float * out_vel, std::vector<Box3D> & det_boxes3d,
  hipStream_t stream)
{
  dim3 blocks(Config::down_grid_size_y);
  dim3 threads(Config::down_grid_size_x);
  generateBoxes3D_kernel<<<blocks, threads, 0, stream>>>(
    out_heatmap, out_offset, out_z, out_dim, out_rot, out_vel, Config::voxel_size_x,
    Config::voxel_size_y, Config::range_min_x, Config::range_min_y, Config::down_grid_size_x,
    Config::down_grid_size_y, Config::downsample_factor, num_class_, score_threshold_,
    thrust::raw_pointer_cast(boxes3d_d_.data()));

  // suppress by socre
  const int num_det_boxes3d =
    thrust::count_if(thrust::device, boxes3d_d_.begin(), boxes3d_d_.end(), is_not_suppressed());
  thrust::device_vector<Box3D> det_boxes3d_d(num_det_boxes3d);
  thrust::copy_if(
    thrust::device, boxes3d_d_.begin(), boxes3d_d_.end(), det_boxes3d_d.begin(),
    is_not_suppressed());

  // sort boxes by score
  thrust::sort(det_boxes3d_d.begin(), det_boxes3d_d.end(), score_greater());

  // supress by NMS
  thrust::device_vector<bool> final_keep_mask_d(num_det_boxes3d);
  const int num_final_det_boxes3d = circleNMS(det_boxes3d_d, final_keep_mask_d, dist_threshold_);

  thrust::device_vector<Box3D> final_det_boxes3d_d(num_final_det_boxes3d);
  thrust::copy_if(
    thrust::device, det_boxes3d_d.begin(), det_boxes3d_d.end(), final_keep_mask_d.begin(),
    final_det_boxes3d_d.begin(), is_kept());

  // device to host
  det_boxes3d.resize(num_final_det_boxes3d);
  thrust::copy(final_det_boxes3d_d.begin(), final_det_boxes3d_d.end(), det_boxes3d.begin());

  return hipGetLastError();
}

}  // namespace centerpoint
